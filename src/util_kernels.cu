#include "hip/hip_runtime.h"
#include "util_kernels.hpp"
#define MAX_GRID_DIM 65535
#include <type_traits> // Required for std::is_floating_point

typedef struct
{
    int y, z;
} grid_factors_t;

template <typename T_complex, int TILE_SIZE, int EPT>
__global__ void swap_axes_kernel(
    T_complex *out,
    const T_complex *in,
    int np0,
    int np1,
    int np2,
    int fold_y,
    int fold_z)
{
    // The shared memory tile now uses the templated complex type
    __shared__ T_complex tile[TILE_SIZE][TILE_SIZE + 1];

    size_t logical_block_x = blockIdx.x;
    size_t extra_y = 0, extra_z = 0;
    if (fold_y > 1)
    {
        extra_y = logical_block_x % fold_y;
        logical_block_x /= fold_y;
    }
    if (fold_z > 1)
    {
        extra_z = logical_block_x % fold_z;
        logical_block_x /= fold_z;
    }
    size_t bx = logical_block_x;
    size_t by = blockIdx.y + extra_y * gridDim.y;
    size_t bz = blockIdx.z + extra_z * gridDim.z;

    size_t lx = threadIdx.x, ly = threadIdx.y;
    size_t y = bz;

// Input: Each thread loads EPT elements along z_in
#pragma unroll
    for (int e = 0; e < EPT; ++e)
    {
        size_t z_in = ly + e * (TILE_SIZE / EPT) + TILE_SIZE * by;
        size_t x_in = lx + TILE_SIZE * bx;
        size_t ind_in = x_in + (y + z_in * (size_t)np1) * (size_t)np0;
        if (x_in < (size_t)np0 && z_in < (size_t)np2 && y < (size_t)np1)
        {
            tile[lx][ly + e * (TILE_SIZE / EPT)] = in[ind_in];
        }
    }

    __syncthreads();

// Output: Each thread writes EPT elements along x_out
#pragma unroll
    for (int e = 0; e < EPT; ++e)
    {
        size_t x_out = ly + e * (TILE_SIZE / EPT) + TILE_SIZE * bx;
        size_t z_out = lx + TILE_SIZE * by;
        size_t ind_out = z_out + (y + x_out * (size_t)np1) * (size_t)np2;
        if (z_out < (size_t)np2 && x_out < (size_t)np0 && y < (size_t)np1)
        {
            out[ind_out] = tile[ly + e * (TILE_SIZE / EPT)][lx];
        }
    }
}

//============================================================================//
//                      HOST LAUNCHER IMPLEMENTATION                          //
//============================================================================//

// This helper function does not depend on the data type and can remain unchanged.
static void set_grid_dims(const int *size,
                          int d2,
                          dim3 *block_dims,
                          dim3 *grid_dims,
                          int elements_per_thread,
                          int tile_size,
                          grid_factors_t *fold_factors)
{
    block_dims->x = tile_size;
    block_dims->y = tile_size / elements_per_thread;
    block_dims->z = 1;

    int nblocks_x = (size[0] + tile_size - 1) / tile_size;
    if (d2 == 0)
        d2 = 1;
    int nblocks_y = (size[d2] + tile_size - 1) / tile_size;
    int nblocks_z = size[(d2 == 1) ? 2 : 1];

    int fold_y = 1, fold_z = 1;
    if (nblocks_y > MAX_GRID_DIM)
    {
        fold_y = (nblocks_y + MAX_GRID_DIM - 1) / MAX_GRID_DIM;
        nblocks_x *= fold_y;
        nblocks_y = MAX_GRID_DIM;
    }
    if (nblocks_z > MAX_GRID_DIM)
    {
        fold_z = (nblocks_z + MAX_GRID_DIM - 1) / MAX_GRID_DIM;
        nblocks_x *= fold_z;
        nblocks_z = MAX_GRID_DIM;
    }
    grid_dims->x = nblocks_x;
    grid_dims->y = nblocks_y;
    grid_dims->z = nblocks_z;

    if (fold_factors)
    {
        fold_factors->y = fold_y;
        fold_factors->z = fold_z;
    }
}

// The host function is now templated on T_complex
template<typename T_complex>
void UtilKernels::swap_axes_cutranspose(const T_complex *d_in, T_complex *d_out,
                                        const unsigned int num_cols, const unsigned int num_rows,
                                        const unsigned int block_size, hipStream_t s)
{
    int sz[3] = {(int)block_size, (int)num_cols, (int)num_rows};
    
    constexpr int EPT = 2;
    constexpr int TILE_SIZE = 32;
    
    dim3 block_dims, grid_dims;
    grid_factors_t fold_factors = {1, 1};

    set_grid_dims(sz, 2, &block_dims, &grid_dims, EPT, TILE_SIZE, &fold_factors);

    // The kernel call itself remains the same
    swap_axes_kernel<T_complex, TILE_SIZE, EPT><<<grid_dims, block_dims, 0, s>>>(
        d_out, d_in, sz[0], sz[1], sz[2], fold_factors.y, fold_factors.z);

    gpuErrchk(hipPeekAtLastError());
}

// --- The explicit instantiations now match the simplified signature ---
template void UtilKernels::swap_axes_cutranspose<ComplexF>(
    const ComplexF*, ComplexF*, unsigned int, unsigned int, unsigned int, hipStream_t);

template void UtilKernels::swap_axes_cutranspose<ComplexD>(
    const ComplexD*, ComplexD*, unsigned int, unsigned int, unsigned int, hipStream_t);

//============================================================================//
//                  CASTING HELPERS (for internal kernel use)                 //
//============================================================================//

// Helper for casting primitive types (float, double)
template <typename T_in, typename T_out>
__device__ __forceinline__ void perform_cast(const T_in &in, T_out &out, std::true_type /* is_floating_point */)
{
    out = in;
}

// Helper for casting complex struct types
template <typename T_in, typename T_out>
__device__ __forceinline__ void perform_cast(const T_in &in, T_out &out, std::false_type /* is_not_floating_point */)
{
    out.x = in.x;
    out.y = in.y;
}

//============================================================================//
//            GENERIC KERNEL IMPLEMENTATIONS (Input and Output Types)         //
//============================================================================//

template <typename T_in, typename T_out>
__global__ void cast_kernel(const T_in *d_in, T_out *d_out, const unsigned int size)
{
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size; idx += gridDim.x * blockDim.x)
    {
        perform_cast(d_in[idx], d_out[idx], typename std::is_floating_point<T_in>::type());
    }
}

template <typename T_in, typename T_out>
__global__ void pad_vector_kernel(const T_in *d_in, T_out *d_pad, const unsigned int padded_size)
{
    const size_t unpadded_size = padded_size / 2;
    const T_in *block_in_base = d_in + (size_t)blockIdx.x * unpadded_size;
    T_out *block_pad_base = d_pad + (size_t)blockIdx.x * padded_size;

    for (size_t j = threadIdx.x; j < padded_size; j += blockDim.x)
    {
        if (j < unpadded_size)
        {
            // Perform the combined copy and cast operation
            perform_cast(block_in_base[j], block_pad_base[j], typename std::is_floating_point<T_in>::type());
        }
        else
        {
            // Zero out the padding using the default constructor T_out() -> 0.0 or {0.0, 0.0}
            block_pad_base[j] = T_out();
        }
    }
}

template <typename T_in, typename T_out>
__global__ void unpad_vector_kernel(const T_in *d_in, T_out *d_unpad, const unsigned int padded_size)
{
    const size_t unpadded_size = padded_size / 2;
    const T_in *block_in_base = d_in + (size_t)blockIdx.x * padded_size;
    T_out *block_unpad_base = d_unpad + (size_t)blockIdx.x * unpadded_size;

    for (size_t j = threadIdx.x; j < unpadded_size; j += blockDim.x)
    {
        // Perform the combined copy and cast operation
        perform_cast(block_in_base[j], block_unpad_base[j], typename std::is_floating_point<T_in>::type());
    }
}

template <typename T_in, typename T_out>
__global__ void repad_vector_kernel(const T_in *d_in, T_out *d_out, const unsigned int padded_size)
{
    const size_t unpadded_size = padded_size / 2;
    const T_in *block_in_base = d_in + (size_t)blockIdx.x * padded_size;
    T_out *block_out_base = d_out + (size_t)blockIdx.x * padded_size;

    for (size_t j = threadIdx.x; j < padded_size; j += blockDim.x)
    {
        if (j < unpadded_size)
        {
            perform_cast(block_in_base[j], block_out_base[j], typename std::is_floating_point<T_in>::type());
        }
        else
        {
            block_out_base[j] = T_out();
        }
    }

    __syncthreads();

    if (threadIdx.x == 0 && (padded_size % 2 == 1))
    {
        size_t nyquist_real_idx = padded_size / 2;
        if (nyquist_real_idx + 1 < padded_size)
        {
            block_out_base[nyquist_real_idx + 1] = T_out();
        }
    }
}

//============================================================================//
//                      HOST LAUNCHER IMPLEMENTATIONS                         //
//============================================================================//

template <typename T_in, typename T_out>
void UtilKernels::cast_vector(const T_in *const d_in, T_out *const d_out, const unsigned int size, hipStream_t s)
{
    if (size == 0)
        return;
    cast_kernel<T_in, T_out><<<(size + 255) / 256, 256, 0, s>>>(d_in, d_out, size);
    gpuErrchk(hipPeekAtLastError());
}

template <typename T_in, typename T_out>
void UtilKernels::pad_vector(const T_in *const d_in, T_out *const d_pad, const unsigned int num_blocks,
                             const unsigned int padded_size, hipStream_t s)
{
    if (padded_size == 0)
        return;
    pad_vector_kernel<T_in, T_out><<<num_blocks, MAX_BLOCK_SIZE, 0, s>>>(d_in, d_pad, padded_size);
    gpuErrchk(hipPeekAtLastError());
}

template <typename T_in, typename T_out>
void unpad_vector(const T_in *const d_in, T_out *const d_unpad, const unsigned int num_blocks,
                  const unsigned int padded_size, hipStream_t s)
{
    if (padded_size == 0)
        return;
    unpad_vector_kernel<T_in, T_out><<<num_blocks, MAX_BLOCK_SIZE, 0, s>>>(d_in, d_unpad, padded_size);
    gpuErrchk(hipPeekAtLastError());
}

template <typename T_in, typename T_out>
void repad_vector(const T_in *const d_in, T_out *const d_repad,
                  const unsigned int num_blocks, const unsigned int padded_size, hipStream_t s)
{
    if (padded_size == 0)
        return;
    repad_vector_kernel<T_in, T_out><<<num_blocks, MAX_BLOCK_SIZE, 0, s>>>(d_in, d_repad, padded_size);
    gpuErrchk(hipPeekAtLastError());
}

template <typename T_in, typename T_out>
void UtilKernels::unpad_repad_vector(const T_in *const d_in, T_out *const d_out,
                                     const unsigned int num_blocks, const unsigned int padded_size,
                                     const bool unpad, hipStream_t s)
{
    if (unpad)
    {
        unpad_vector<T_in, T_out>(d_in, d_out, num_blocks, padded_size, s);
    }
    else
    {
        repad_vector<T_in, T_out>(d_in, d_out, num_blocks, padded_size, s);
    }
}

//============================================================================//
//                      EXPLICIT TEMPLATE INSTANTIATIONS                      //
//============================================================================//

// --- cast_vector: For pure precision changes of the same type category ---
template void UtilKernels::cast_vector<float, double>(const float *, double *, unsigned int, hipStream_t);
template void UtilKernels::cast_vector<double, float>(const double *, float *, unsigned int, hipStream_t);
template void UtilKernels::cast_vector<ComplexF, ComplexD>(const ComplexF *, ComplexD *, unsigned int, hipStream_t);
template void UtilKernels::cast_vector<ComplexD, ComplexF>(const ComplexD *, ComplexF *, unsigned int, hipStream_t);

// --- pad_vector: For padding REAL -> REAL, with optional precision change ---
template void UtilKernels::pad_vector<float, float>(const float *, float *, unsigned int, unsigned int, hipStream_t);
template void UtilKernels::pad_vector<double, double>(const double *, double *, unsigned int, unsigned int, hipStream_t);
template void UtilKernels::pad_vector<float, double>(const float *, double *, unsigned int, unsigned int, hipStream_t);
template void UtilKernels::pad_vector<double, float>(const double *, float *, unsigned int, unsigned int, hipStream_t);

// Unpad

template void UtilKernels::unpad_repad_vector<float, float>(const float *, float *, unsigned int, unsigned int, bool, hipStream_t);
template void UtilKernels::unpad_repad_vector<double, double>(const double *, double *, unsigned int, unsigned int, bool, hipStream_t);
template void UtilKernels::unpad_repad_vector<float, double>(const float *, double *, unsigned int, unsigned int, bool, hipStream_t);
template void UtilKernels::unpad_repad_vector<double, float>(const double *, float *, unsigned int, unsigned int, bool, hipStream_t);